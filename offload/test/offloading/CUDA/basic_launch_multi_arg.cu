// clang-format off
// RUN: %clang++ %flags -foffload-via-llvm --offload-arch=native %s -o %t
// RUN: %t | %fcheck-generic
// RUN: %clang++ %flags -foffload-via-llvm --offload-arch=native %s -o %t -fopenmp 
// RUN: %t | %fcheck-generic
// clang-format on

// UNSUPPORTED: aarch64-unknown-linux-gnu
// UNSUPPORTED: aarch64-unknown-linux-gnu-LTO
// UNSUPPORTED: x86_64-unknown-linux-gnu
// UNSUPPORTED: x86_64-unknown-linux-gnu-LTO


#include <hip/hip_runtime.h>
#include <stdio.h>

#include "hip/hip_runtime_api.h"

__global__ void square(int *Dst, short Q, int *Src, short P) {
  *Dst = (Src[0] + Src[1]) * (Q + P);
  Src[0] = Q;
  Src[1] = P;
}

int main(int argc, char **argv) {
  int DevNo = 0;
  int *Src, *Ptr;
  hipMalloc(&Ptr, 4);
  hipMalloc(&Src, 8);

  int I = 7;
  int HostSrc[2] = {-2,8};
  hipMemcpy(Ptr, &I, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Src, &HostSrc[0], 2*sizeof(int), hipMemcpyHostToDevice);
  square<<<1, 1>>>(Ptr, 3, Src, 4);
  hipMemcpy(&I, Ptr, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&HostSrc[0], Src, 2 * sizeof(int), hipMemcpyDeviceToHost);
  printf("I: %i\n", I);
  // CHECK: I: 42
  printf("Src: %i, %i\n", I);
  // CHECK: Src: 3, 4
}
